#include "hip/hip_runtime.h"
//
// Copyright 2010-2011,2014 Ettus Research LLC
// Copyright 2018 Ettus Research, a National Instruments Company
//
// SPDX-License-Identifier: GPL-3.0-or-later
//

#include <uhd/types/tune_request.hpp>
#include <uhd/utils/thread.hpp>
#include <uhd/utils/safe_main.hpp>
#include <uhd/usrp/multi_usrp.hpp>
#include <uhd/exception.hpp>
#include <boost/program_options.hpp>
#include <boost/format.hpp>
#include <boost/lexical_cast.hpp>
#include <boost/circular_buffer.hpp>
#include <boost/thread.hpp>
#include <iostream>
#include <fstream>
#include <csignal>
#include <complex>
#include <thread>
#include <chrono>

// includes, project
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <hipfft/hipfft.h>

namespace po = boost::program_options;

static bool stop_signal_called = false;
void sig_int_handler(int){stop_signal_called = true;}

#define num_buffers 384

typedef std::complex<float> sample_t;

typedef struct mdbuff
{
    std::vector<sample_t> data;
    uhd::rx_metadata_t    md;
} mdbuff_t;

__global__ void calcPow(hipFloatComplex *a, int N)
{
    int n   = N/blockDim.x;
    int idx = threadIdx.x*n;
    for (int i = idx; i < idx+n; i++) {
        a[i] = hipCmulf(a[i], hipConjf(a[i]));
    }
}

__global__ void calcSum(float *d, float *a, int N)
{
    int n   = N/blockDim.x;
    int idx = threadIdx.x*n;
    for (int i = idx; i < idx+n; i++) {
        a[i] += d[i*2];
    }
}

__global__ void calcMax(float *d, float *x, int N)
{
    int n   = N/blockDim.x;
    int idx = threadIdx.x*n;
    for (int i = idx; i < idx+n; i++) {
        x[i] = fmaxf(x[i], d[i*2]);
    }
}

__global__ void normalize(float *d, int N, float factor)
{
    int n   = N/blockDim.x;
    int idx = threadIdx.x*n;
    for (int i = idx; i < idx+n; i++) {
        d[i] /= factor;
    }
}

void cuda_handler(boost::circular_buffer<mdbuff_t *> *cb, uint32_t bufsz, uint32_t accum)
{
    hipFloatComplex *d_i; 
    float          *d_a; 
    float          *d_x; 
    hipfftHandle plan;
    mdbuff_t *buf;

    float *h_a;
    float *h_x;

    uint32_t cnt = 0;

    FILE *ofile = fopen("test.dat", "wb"); //remove

    // Allocate output buffers
    h_a = (float *)malloc(sizeof(float) * bufsz);
    h_x = (float *)malloc(sizeof(float) * bufsz);

    // Allocate the input buffer
    hipMalloc((void **)(&d_i), sizeof(hipFloatComplex) * bufsz);

    // Allocate the accumulator
    hipMalloc((void **)(&d_a), sizeof(float) * bufsz);
    hipMemset(d_a, 0, sizeof(float) * bufsz);

    // Allocate the peak detector
    hipMalloc((void **)(&d_x), sizeof(float) * bufsz);
    hipMemset(d_x, 0, sizeof(float) * bufsz);

    // Generate the FFT plan.
    hipfftPlan1d(&plan, bufsz, HIPFFT_C2C, 1);

    // Loop till stopped.
    while (!stop_signal_called) 
    {
        // Avoid tight-looping the processor. 
        while (cb->empty())
        {
            boost::this_thread::sleep_for(boost::chrono::milliseconds(10));
        }

        // Copy the data up to the GPU and pop it off the circular buffer
        buf = (*cb)[0];
        std::cout << buf->md.to_pp_string() << std::endl;
        hipMemcpy(d_i, &buf->data.front(), sizeof(hipFloatComplex) * bufsz, hipMemcpyHostToDevice);
        cb->pop_front();

        // Run the FFT plan in-place (clobber input data).
        hipfftExecC2C(plan, (hipfftComplex *)(d_i), (hipfftComplex *)(d_i), HIPFFT_FORWARD);

        // Compute the mag squared in-place.
        calcPow<<<1, 512>>>(d_i, bufsz);

        // Sum to the accumulator
        calcSum<<<1, 512>>>((float *)d_i, d_a, bufsz);

        // Compare with max.
        calcMax<<<1, 512>>>((float *)d_i, d_x, bufsz);

        // If we've accumulated enough frames, copy back.
        if (cnt == accum)
        {
            // Average
            normalize<<<1, 512>>>(d_a, bufsz, (float)accum);

            hipMemcpy(h_a, d_a, sizeof(float) * bufsz, hipMemcpyDeviceToHost);
            hipMemcpy(h_x, d_x, sizeof(float) * bufsz, hipMemcpyDeviceToHost);

            hipMemset(d_a, 0, sizeof(float) * bufsz);
            hipMemset(d_x, 0, sizeof(float) * bufsz);

            cnt = 0;
            fwrite(h_a, sizeof(float), bufsz, ofile);
            fwrite(h_x, sizeof(float), bufsz, ofile);

            std::cout << "." << std::flush;
        }
        else
        {
            cnt += 1;
        }
    }

    free(h_a);
    free(h_x);
    fclose(ofile);
};


void recv_to_file(
    uhd::usrp::multi_usrp::sptr usrp,
    const std::string &cpu_format,
    const std::string &wire_format,
    const std::string &channel,
    size_t samps_per_buff
)
{
    mdbuff_t buffs[num_buffers];
    //std::vector<void *> bufflist;

    for (uint32_t i = 0; i < num_buffers; i++)
    {
        buffs[i].data.resize(samps_per_buff);
        //bufflist.push_back(&buffs[i].front());
    }

    boost::circular_buffer<mdbuff_t *> cb(num_buffers);
    boost::thread gpu_thread(cuda_handler, &cb, samps_per_buff, 1000);
    uint32_t bufidx = 0;
    uint32_t overrun_cnt = 0;

    //create a receive streamer
    uhd::stream_args_t stream_args(cpu_format,wire_format);
    std::vector<size_t> channel_nums;
    channel_nums.push_back(boost::lexical_cast<size_t>(channel));
    stream_args.channels = channel_nums;
    uhd::rx_streamer::sptr rx_stream = usrp->get_rx_stream(stream_args);

    //std::vector<samp_type> buff(samps_per_buff);
    bool overflow_message = true;

    //setup streaming
    uhd::stream_cmd_t stream_cmd(uhd::stream_cmd_t::STREAM_MODE_START_CONTINUOUS);
    stream_cmd.num_samps = size_t(0);
    stream_cmd.stream_now = true;
    stream_cmd.time_spec = uhd::time_spec_t();
    rx_stream->issue_stream_cmd(stream_cmd);

    // Run this loop until either time expired (if a duration was given), until
    // the requested number of samples were collected (if such a number was
    // given), or until Ctrl-C was pressed.
    while (not stop_signal_called) 
    {

        size_t num_rx_samps = rx_stream->recv(&buffs[bufidx].data.front(), buffs[bufidx].data.size(), buffs[bufidx].md, 3.0, false);

        if (buffs[bufidx].md.error_code == uhd::rx_metadata_t::ERROR_CODE_TIMEOUT)
        {
            std::cout << boost::format("Timeout while streaming") << std::endl;
            break;
        }

        if (buffs[bufidx].md.error_code == uhd::rx_metadata_t::ERROR_CODE_OVERFLOW)
        {
            if (overflow_message) 
            {
                overflow_message = false;
                std::cerr << boost::format(
                    "Got an overflow indication. Please consider the following:\n"
                    "  Your write medium must sustain a rate of %fMB/s.\n"
                    "  Dropped samples will not be written to the file.\n"
                    "  Please modify this example for your purposes.\n"
                    "  This message will not appear again.\n"
                ) % (usrp->get_rx_rate()*sizeof(sample_t)/1e6);
            }
            continue;
        }

        if (buffs[bufidx].md.error_code != uhd::rx_metadata_t::ERROR_CODE_NONE)
        {
            std::string error = str(boost::format("Receiver error: %s") % buffs[bufidx].md.strerror());
            std::cerr << error << std::endl;
            continue;
        }

        if (cb.full())
        {
            std::cerr << ++overrun_cnt << " Overrun(s)." << std::endl;
        }

        cb.push_back(&buffs[bufidx]);
        bufidx = (bufidx + 1) % num_buffers;

    }

    stream_cmd.stream_mode = uhd::stream_cmd_t::STREAM_MODE_STOP_CONTINUOUS;
    rx_stream->issue_stream_cmd(stream_cmd);

    std::cout << "Waiting for GPU thread... ";
    gpu_thread.join();
    std::cout << "Done." << std::endl;

}

typedef std::function<uhd::sensor_value_t(const std::string&)> get_sensor_fn_t;

bool check_locked_sensor(
    std::vector<std::string> sensor_names,
    const char* sensor_name,
    get_sensor_fn_t get_sensor_fn,
    double setup_time
) {
    if (std::find(sensor_names.begin(), sensor_names.end(), sensor_name) == sensor_names.end())
        return false;

    auto setup_timeout =
        std::chrono::steady_clock::now()
        + std::chrono::milliseconds(int64_t(setup_time * 1000));
    bool lock_detected = false;

    std::cout << boost::format("Waiting for \"%s\": ") % sensor_name;
    std::cout.flush();

    while (true) {
        if (lock_detected and
            (std::chrono::steady_clock::now() > setup_timeout)) {
            std::cout << " locked." << std::endl;
            break;
        }
        if (get_sensor_fn(sensor_name).to_bool()) {
            std::cout << "+";
            std::cout.flush();
            lock_detected = true;
        }
        else {
            if (std::chrono::steady_clock::now() > setup_timeout) {
                std::cout << std::endl;
                throw std::runtime_error(str(
                    boost::format("timed out waiting for consecutive locks on sensor \"%s\"")
                    % sensor_name
                ));
            }
            std::cout << "_";
            std::cout.flush();
        }
        std::this_thread::sleep_for(std::chrono::milliseconds(100));
    }
    std::cout << std::endl;
    return true;
}

int UHD_SAFE_MAIN(int argc, char *argv[])
{
    uhd::set_thread_priority_safe();

    //variables to be set by po
    std::string args, type, ant, subdev, ref, wirefmt, channel;
    size_t spb;
    double rate, freq, gain, setup_time;

    //setup the program options
    po::options_description desc("Allowed options");
    desc.add_options()
        ("help", "help message")
        ("args", po::value<std::string>(&args)->default_value(""), "multi uhd device address args")
        ("spb", po::value<size_t>(&spb)->default_value((1<<20)), "samples per buffer")
        ("rate", po::value<double>(&rate)->default_value(25e6), "rate of incoming samples")
        ("freq", po::value<double>(&freq)->default_value(200e6), "RF center frequency in Hz")
        ("gain", po::value<double>(&gain), "gain for the RF chain")
        ("ant", po::value<std::string>(&ant), "antenna selection")
        ("subdev", po::value<std::string>(&subdev), "subdevice specification")
        ("channel", po::value<std::string>(&channel)->default_value("0"), "which channel to use")
        ("ref", po::value<std::string>(&ref)->default_value("internal"), "reference source (internal, external, mimo)")
        ("wirefmt", po::value<std::string>(&wirefmt)->default_value("sc16"), "wire format (sc8, sc16 or s16)")
        ("setup", po::value<double>(&setup_time)->default_value(1.0), "seconds of setup time")
        ("skip-lo", "skip checking LO lock status")
        ("int-n", "tune USRP with integer-N tuning")
    ;
    po::variables_map vm;
    po::store(po::parse_command_line(argc, argv, desc), vm);
    po::notify(vm);

    //print the help message
    if (vm.count("help")) {
        std::cout << boost::format("UHD RX samples to file %s") % desc << std::endl;
        std::cout
            << std::endl
            << "This application streams data from a single channel of a USRP device to a file.\n"
            << std::endl;
        return ~0;
    }

    //create a usrp device
    std::cout << std::endl;
    std::cout << boost::format("Creating the usrp device with: %s...") % args << std::endl;
    uhd::usrp::multi_usrp::sptr usrp = uhd::usrp::multi_usrp::make(args);

    //Lock mboard clocks
    usrp->set_clock_source(ref);

    //always select the subdevice first, the channel mapping affects the other settings
    if (vm.count("subdev")) usrp->set_rx_subdev_spec(subdev);

    std::cout << boost::format("Using Device: %s") % usrp->get_pp_string() << std::endl;

    //set the sample rate
    if (rate <= 0.0){
        std::cerr << "Please specify a valid sample rate" << std::endl;
        return ~0;
    }
    std::cout << boost::format("Setting RX Rate: %f Msps...") % (rate/1e6) << std::endl;
    usrp->set_rx_rate(rate);
    std::cout << boost::format("Actual RX Rate: %f Msps...") % (usrp->get_rx_rate()/1e6) << std::endl << std::endl;

    //set the center frequency
    if (vm.count("freq")) { //with default of 0.0 this will always be true
        std::cout << boost::format("Setting RX Freq: %f MHz...") % (freq/1e6) << std::endl;
        uhd::tune_request_t tune_request(freq);
        if(vm.count("int-n")) tune_request.args = uhd::device_addr_t("mode_n=integer");
        usrp->set_rx_freq(tune_request);
        std::cout << boost::format("Actual RX Freq: %f MHz...") % (usrp->get_rx_freq()/1e6) << std::endl << std::endl;
    }

    //set the rf gain
    if (vm.count("gain")) {
        std::cout << boost::format("Setting RX Gain: %f dB...") % gain << std::endl;
        usrp->set_rx_gain(gain);
        std::cout << boost::format("Actual RX Gain: %f dB...") % usrp->get_rx_gain() << std::endl << std::endl;
    }

    //set the antenna
    if (vm.count("ant")) usrp->set_rx_antenna(ant);

    std::this_thread::sleep_for(
        std::chrono::milliseconds(int64_t(1000 * setup_time))
    );

    //check Ref and LO Lock detect
    if (not vm.count("skip-lo")){
        check_locked_sensor(
            usrp->get_rx_sensor_names(0),
            "lo_locked",
            [usrp](const std::string& sensor_name){
                return usrp->get_rx_sensor(sensor_name);
            },
            setup_time
        );
        if (ref == "mimo") {
            check_locked_sensor(
                usrp->get_mboard_sensor_names(0),
                "mimo_locked",
                [usrp](const std::string& sensor_name){
                    return usrp->get_mboard_sensor(sensor_name);
                },
                setup_time
            );
        }
        if (ref == "external") {
            check_locked_sensor(
                usrp->get_mboard_sensor_names(0),
                "ref_locked",
                [usrp](const std::string& sensor_name){
                    return usrp->get_mboard_sensor(sensor_name);
                },
                setup_time
            );
        }
    }

    std::signal(SIGINT, &sig_int_handler);
    std::cout << "Press Ctrl + C to stop streaming..." << std::endl;

    //recv to file
    recv_to_file(usrp, "fc32", wirefmt, channel, spb);

    //finished
    std::cout << std::endl << "Done!" << std::endl << std::endl;

    return EXIT_SUCCESS;
}
